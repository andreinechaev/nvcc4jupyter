
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <set>
#include <string>
#include <iterator>

#include <tuple>

struct S {
    int n;
    std::string s;
    float d;
    bool operator<(const S& rhs) const
    {
        // compares n to rhs.n,
        // then s to rhs.s,
        // then d to rhs.d
        return std::tie(n, s, d) < std::tie(rhs.n, rhs.s, rhs.d);
    }
};

int main()
{
    std::set<S> mySet;

    // pre C++17:
    {
	    S value{42, "Test", 3.14};
	    std::set<S>::iterator iter;
	    bool inserted;

	    // unpacks the return val of insert into iter and inserted
	    std::tie(iter, inserted) = mySet.insert(value);

	    if (inserted)
		    std::cout << "Value was inserted\n";
    }

	// with C++17:
    {
        S value{100, "abc", 100.0};
        const auto [iter, inserted] = mySet.insert(value);

        if (inserted)
		    std::cout << "Value(" << iter->n << ", " << iter->s << ", ...) was inserted" << "\n";
    }
}
